#include "hip/hip_runtime.h"
#include "mc_data.cuh"



__device__ static inline float get_sdf(uint3 bpos, uint arx, uint ary, uint arz,
                                              const IndexerAccessor indexer,
                                              const CubeSDFAccessor cube_sdf
                                              ) {
    uint r = cube_sdf.size(1);
    if (arx >= r) { bpos.x += 1; arx = 0; }
    // if (bpos.x >= bsize.x) { bpos.x = bsize.x - 1; rpos.x = r - 1; }
    if (ary >= r) { bpos.y += 1; ary = 0; }
    // if (bpos.y >= bsize.y) { bpos.y = bsize.y - 1; rpos.y = r - 1; }
    if (arz >= r) { bpos.z += 1; arz = 0; }
    // if (bpos.z >= bsize.z) { bpos.z = bsize.z - 1; rpos.z = r - 1; }
    if (bpos.x >= indexer.size(0) || bpos.y >= indexer.size(1) || bpos.z >= indexer.size(2)) {
        return NAN;
    }
//    printf("B-Getting: %d %d %d --> %d, %d, %d\n", bx, by, bz, indexer.size(0), indexer.size(1), indexer.size(2));
    int batch_ind = indexer[bpos.x][bpos.y][bpos.z];
    if (batch_ind == -1) {
        return NAN;
    }
//    printf("Getting: %d %d %d %d --> %d %d\n", batch_ind, arx, ary, arz, cube_sdf.size(0), cube_sdf.size(1));
    float sdf = cube_sdf[batch_ind][arx][ary][arz];

    return sdf;

}


__device__ static inline float3 sdf_interp(const float3 p1, const float3 p2, float valp1, float valp2) {
    if (fabs(0.0f - valp1) < 1.0e-5f) return p1;
	if (fabs(0.0f - valp2) < 1.0e-5f) return p2;
	if (fabs(valp1 - valp2) < 1.0e-5f) return p1;

	float w2 = (0.0f - valp1) / (valp2 - valp1);
	float w1 = 1 - w2;

	return make_float3(p1.x * w1 + p2.x * w2,
	                   p1.y * w1 + p2.y * w2,
	                   p1.z * w1 + p2.z * w2);
}

__global__ static void meshing_cube(const ValidBlocksAccessor valid_blocks,
                                    const IndexerAccessor indexer,
                                    const CubeSDFAccessor cube_sdf,
                                    TrianglesAccessor triangles,
                                    TriangleVecIdAccessor triangle_flatten_id,
                                    int* __restrict__ triangles_count,
                                    int max_triangles_count,
                                    int nx, int ny, int nz
                                    ) {
    const uint r = cube_sdf.size(1);
    const uint r3 = r * r * r;
    const uint num_lif = valid_blocks.size(0);
    const float sbs = 1.0f / r;         // sub-block-size

    const uint lif_id = blockIdx.x * blockDim.x + threadIdx.x;
    const uint sub_id = blockIdx.y * blockDim.y + threadIdx.y;

    if (lif_id >= num_lif || sub_id >= r3) {
        return;
    }

    const uint3 bpos = make_uint3(
        (valid_blocks[lif_id] / (ny * nz)) % nx,
        (valid_blocks[lif_id] / nz) % ny,
        valid_blocks[lif_id] % nz);
    const uint3 bsize = make_uint3(indexer.size(0), indexer.size(1), indexer.size(2));
    const uint rx = sub_id / (r * r);
    const uint ry = (sub_id / r) % r;
    const uint rz = sub_id % r;

    // Find all 8 neighbours
    float3 points[8];
    float sdf_vals[8];

    sdf_vals[0] = get_sdf(bpos, rx, ry, rz, indexer, cube_sdf);
    if (isnan(sdf_vals[0])) return;
    points[0] = make_float3(bpos.x + rx * sbs, bpos.y + ry * sbs, bpos.z + rz * sbs);

    sdf_vals[1] = get_sdf(bpos, rx + 1, ry, rz, indexer, cube_sdf);
    if (isnan(sdf_vals[1])) return;
    points[1] = make_float3(bpos.x + (rx + 1) * sbs, bpos.y + ry * sbs, bpos.z + rz * sbs);

    sdf_vals[2] = get_sdf(bpos,rx + 1, ry + 1, rz, indexer, cube_sdf);
    if (isnan(sdf_vals[2])) return;
    points[2] = make_float3(bpos.x + (rx + 1) * sbs, bpos.y + (ry + 1) * sbs, bpos.z + rz * sbs);

    sdf_vals[3] = get_sdf(bpos, rx, ry + 1, rz, indexer, cube_sdf);
    if (isnan(sdf_vals[3])) return;
    points[3] = make_float3(bpos.x + rx * sbs, bpos.y + (ry + 1) * sbs, bpos.z + rz * sbs);

    sdf_vals[4] = get_sdf(bpos, rx, ry, rz + 1, indexer, cube_sdf);
    if (isnan(sdf_vals[4])) return;
    points[4] = make_float3(bpos.x + rx * sbs, bpos.y + ry * sbs, bpos.z + (rz + 1) * sbs);

    sdf_vals[5] = get_sdf(bpos, rx + 1, ry, rz + 1, indexer, cube_sdf);
    if (isnan(sdf_vals[5])) return;
    points[5] = make_float3(bpos.x + (rx + 1) * sbs, bpos.y + ry * sbs, bpos.z + (rz + 1) * sbs);

    sdf_vals[6] = get_sdf(bpos,rx + 1, ry + 1, rz + 1, indexer, cube_sdf);
    if (isnan(sdf_vals[6])) return;
    points[6] = make_float3(bpos.x + (rx + 1) * sbs, bpos.y + (ry + 1) * sbs, bpos.z + (rz + 1) * sbs);

    sdf_vals[7] = get_sdf(bpos, rx, ry + 1, rz + 1, indexer, cube_sdf);
    if (isnan(sdf_vals[7])) return;
    points[7] = make_float3(bpos.x + rx * sbs, bpos.y + (ry + 1) * sbs, bpos.z + (rz + 1) * sbs);

    // Find triangle config.
    int cube_type = 0;
	if (sdf_vals[0] < 0) cube_type |= 1; if (sdf_vals[1] < 0) cube_type |= 2;
	if (sdf_vals[2] < 0) cube_type |= 4; if (sdf_vals[3] < 0) cube_type |= 8;
	if (sdf_vals[4] < 0) cube_type |= 16; if (sdf_vals[5] < 0) cube_type |= 32;
	if (sdf_vals[6] < 0) cube_type |= 64; if (sdf_vals[7] < 0) cube_type |= 128;

	// Find vertex position on each edge (weighted by sdf value)
	int edge_config = edgeTable[cube_type];
	float3 vert_list[12];

	if (edge_config == 0) return;
    if (edge_config & 1) vert_list[0] = sdf_interp(points[0], points[1], sdf_vals[0], sdf_vals[1]);
	if (edge_config & 2) vert_list[1] = sdf_interp(points[1], points[2], sdf_vals[1], sdf_vals[2]);
	if (edge_config & 4) vert_list[2] = sdf_interp(points[2], points[3], sdf_vals[2], sdf_vals[3]);
	if (edge_config & 8) vert_list[3] = sdf_interp(points[3], points[0], sdf_vals[3], sdf_vals[0]);
	if (edge_config & 16) vert_list[4] = sdf_interp(points[4], points[5], sdf_vals[4], sdf_vals[5]);
	if (edge_config & 32) vert_list[5] = sdf_interp(points[5], points[6], sdf_vals[5], sdf_vals[6]);
	if (edge_config & 64) vert_list[6] = sdf_interp(points[6], points[7], sdf_vals[6], sdf_vals[7]);
	if (edge_config & 128) vert_list[7] = sdf_interp(points[7], points[4], sdf_vals[7], sdf_vals[4]);
	if (edge_config & 256) vert_list[8] = sdf_interp(points[0], points[4], sdf_vals[0], sdf_vals[4]);
	if (edge_config & 512) vert_list[9] = sdf_interp(points[1], points[5], sdf_vals[1], sdf_vals[5]);
	if (edge_config & 1024) vert_list[10] = sdf_interp(points[2], points[6], sdf_vals[2], sdf_vals[6]);
	if (edge_config & 2048) vert_list[11] = sdf_interp(points[3], points[7], sdf_vals[3], sdf_vals[7]);

    float3 vp[3];
    // Write triangles to array.
    for (int i = 0; triangleTable[cube_type][i] != -1; i += 3) {
#pragma unroll
        for (int vi = 0; vi < 3; ++vi) {
            vp[vi] = vert_list[triangleTable[cube_type][i + vi]];
        }
        int triangle_id = atomicAdd(triangles_count, 1);
        if (triangle_id < max_triangles_count) {
#pragma unroll
            for (int vi = 0; vi < 3; ++ vi) {
                triangles[triangle_id][vi][0] = vp[vi].x;
                triangles[triangle_id][vi][1] = vp[vi].y;
                triangles[triangle_id][vi][2] = vp[vi].z;
            }
            triangle_flatten_id[triangle_id] = valid_blocks[lif_id];
        }
    }

}




__inline__ __device__ bool isvalid(uint x,uint y,uint z,const MaskAccessor mask,uint nx,uint ny,uint nz){
    if (x >= nx || y >= ny || z >= nz){
        return false;
    } 
    return mask[x][y][z];
}

__global__ static void meshing_cube_dense(const CordAccessor valid_cords,
                                    const MaskAccessor mask,
                                    const TrianglesAccessor dense_sdf,
                                    TrianglesAccessor triangles,
                                    int* __restrict__ triangles_count,
                                    int max_triangles_count
                                    ) {
    const uint num_lif = valid_cords.size(0);
    const uint cord_id = blockIdx.x * blockDim.x + threadIdx.x;
    const uint nx = mask.size(0),ny = mask.size(1),nz = mask.size(2);
    if (cord_id >= num_lif) {
        return;
    }

    const uint3 bpos = make_uint3(
        valid_cords[cord_id][0],
        valid_cords[cord_id][1],
        valid_cords[cord_id][2]);

    // Find all 8 neighbours
    float3 points[8];
    float sdf_vals[8];

    if(!isvalid(bpos.x,bpos.y,bpos.z,mask,nx,ny,nz)) return;
    sdf_vals[0] = dense_sdf[bpos.x][bpos.y][bpos.z];
    points[0] = make_float3(bpos.x, bpos.y, bpos.z );

    if(!isvalid(bpos.x + 1,bpos.y,bpos.z,mask,nx,ny,nz))return;
    sdf_vals[1] = dense_sdf[bpos.x + 1][bpos.y][bpos.z];
    points[1] = make_float3(bpos.x +1, bpos.y , bpos.z );

    if(!isvalid(bpos.x + 1,bpos.y + 1,bpos.z,mask,nx,ny,nz))return;
    sdf_vals[2] = dense_sdf[bpos.x + 1][bpos.y + 1][bpos.z];
    points[2] = make_float3(bpos.x + 1, bpos.y + 1, bpos.z );

    if(!isvalid(bpos.x,bpos.y + 1,bpos.z,mask,nx,ny,nz))return;
    sdf_vals[3] = dense_sdf[bpos.x][bpos.y + 1][bpos.z];
    points[3] = make_float3(bpos.x , bpos.y + 1, bpos.z);

    if(!isvalid(bpos.x,bpos.y ,bpos.z + 1,mask,nx,ny,nz))return;
    sdf_vals[4] = dense_sdf[bpos.x][bpos.y][bpos.z + 1];
    points[4] = make_float3(bpos.x , bpos.y , bpos.z + 1);

    if(!isvalid(bpos.x + 1,bpos.y ,bpos.z + 1,mask,nx,ny,nz))return;
    sdf_vals[5] = dense_sdf[bpos.x + 1][bpos.y][bpos.z + 1];
    points[5] = make_float3(bpos.x + 1, bpos.y , bpos.z + 1);

    if(!isvalid(bpos.x + 1,bpos.y + 1,bpos.z + 1,mask,nx,ny,nz))return;
    sdf_vals[6] = dense_sdf[bpos.x + 1][bpos.y + 1][bpos.z + 1];
    points[6] = make_float3(bpos.x + 1, bpos.y + 1, bpos.z + 1);
    
    if(!isvalid(bpos.x,bpos.y + 1,bpos.z + 1,mask,nx,ny,nz))return;
    sdf_vals[7] = dense_sdf[bpos.x][bpos.y + 1][bpos.z + 1];
    points[7] = make_float3(bpos.x , bpos.y + 1, bpos.z + 1);
    // printf("%d %d %d\n",bpos.x,bpos.y,bpos.z);

    // Find triangle config.
    int cube_type = 0;
	if (sdf_vals[0] < 0) cube_type |= 1; if (sdf_vals[1] < 0) cube_type |= 2;
	if (sdf_vals[2] < 0) cube_type |= 4; if (sdf_vals[3] < 0) cube_type |= 8;
	if (sdf_vals[4] < 0) cube_type |= 16; if (sdf_vals[5] < 0) cube_type |= 32;
	if (sdf_vals[6] < 0) cube_type |= 64; if (sdf_vals[7] < 0) cube_type |= 128;

	// Find vertex position on each edge (weighted by sdf value)
	int edge_config = edgeTable[cube_type];
	float3 vert_list[12];

	if (edge_config == 0) return;
    if (edge_config & 1) vert_list[0] = sdf_interp(points[0], points[1], sdf_vals[0], sdf_vals[1]);
	if (edge_config & 2) vert_list[1] = sdf_interp(points[1], points[2], sdf_vals[1], sdf_vals[2]);
	if (edge_config & 4) vert_list[2] = sdf_interp(points[2], points[3], sdf_vals[2], sdf_vals[3]);
	if (edge_config & 8) vert_list[3] = sdf_interp(points[3], points[0], sdf_vals[3], sdf_vals[0]);
	if (edge_config & 16) vert_list[4] = sdf_interp(points[4], points[5], sdf_vals[4], sdf_vals[5]);
	if (edge_config & 32) vert_list[5] = sdf_interp(points[5], points[6], sdf_vals[5], sdf_vals[6]);
	if (edge_config & 64) vert_list[6] = sdf_interp(points[6], points[7], sdf_vals[6], sdf_vals[7]);
	if (edge_config & 128) vert_list[7] = sdf_interp(points[7], points[4], sdf_vals[7], sdf_vals[4]);
	if (edge_config & 256) vert_list[8] = sdf_interp(points[0], points[4], sdf_vals[0], sdf_vals[4]);
	if (edge_config & 512) vert_list[9] = sdf_interp(points[1], points[5], sdf_vals[1], sdf_vals[5]);
	if (edge_config & 1024) vert_list[10] = sdf_interp(points[2], points[6], sdf_vals[2], sdf_vals[6]);
	if (edge_config & 2048) vert_list[11] = sdf_interp(points[3], points[7], sdf_vals[3], sdf_vals[7]);
    float3 vp[3];
    // Write triangles to array.
    for (int i = 0; triangleTable[cube_type][i] != -1; i += 3) {
#pragma unroll
        for (int vi = 0; vi < 3; ++vi) {
            vp[vi] = vert_list[triangleTable[cube_type][i + vi]];
        }
        int triangle_id = atomicAdd(triangles_count, 1);
        if (triangle_id < max_triangles_count) {
#pragma unroll
            for (int vi = 0; vi < 3; ++ vi) {
                triangles[triangle_id][vi][0] = vp[vi].x;
                triangles[triangle_id][vi][1] = vp[vi].y;
                triangles[triangle_id][vi][2] = vp[vi].z;
            }
        }
    }

}



std::vector<torch::Tensor> marching_cubes_sparse_cuda(
    torch::Tensor valid_blocks,         // (K, )     
    torch::Tensor batch_indexer,    // (nx,ny,nz) -> batch id
    torch::Tensor cube_sdf,             // (M, rx, ry, rz)
    int max_n_triangles,                // Maximum number of triangle buffer.
    const std::vector<int>& n_xyz    // [nx, ny, nz]
) {
    CHECK_INPUT(valid_blocks);
    CHECK_INPUT(cube_sdf);
    CHECK_INPUT(batch_indexer);
    assert(max_n_triangles > 0);

    const int r = cube_sdf.size(1);
    const int r3 = r * r * r;
    const int num_lif = valid_blocks.size(0);

    torch::Tensor triangles = torch::empty({max_n_triangles, 3, 3},
                                           torch::dtype(torch::kFloat32).device(torch::kCUDA));
    torch::Tensor triangle_flatten_id = torch::empty({max_n_triangles}, torch::dtype(torch::kLong).device(torch::kCUDA));

    dim3 dimBlock = dim3(16, 16);
    uint xBlocks = (num_lif + dimBlock.x - 1) / dimBlock.x;
    uint yBlocks = (r3 + dimBlock.y - 1) / dimBlock.y;
    dim3 dimGrid = dim3(xBlocks, yBlocks);

    thrust::device_vector<int> n_output(1, 0);
    meshing_cube<<<dimGrid, dimBlock>>>(
        valid_blocks.packed_accessor32<int64_t, 1, torch::RestrictPtrTraits>(),
        batch_indexer.packed_accessor32<int64_t, 3, torch::RestrictPtrTraits>(),
        cube_sdf.packed_accessor32<float, 4, torch::RestrictPtrTraits>(),
        triangles.packed_accessor32<float, 3, torch::RestrictPtrTraits>(),
        triangle_flatten_id.packed_accessor32<int64_t, 1, torch::RestrictPtrTraits>(),
        n_output.data().get(), max_n_triangles, 
        n_xyz[0], n_xyz[1], n_xyz[2]
    );

    int output_n_triangles = n_output[0];
    if (output_n_triangles < max_n_triangles) {
        // Trim output tensor if it is not full.
        triangles = triangles.index({torch::indexing::Slice(torch::indexing::None, output_n_triangles)});
        triangle_flatten_id = triangle_flatten_id.index({torch::indexing::Slice(torch::indexing::None, output_n_triangles)});
    } else {
        // Otherwise spawn a warning.
        std::cerr << "Warning from marching cube: the max triangle number is too small " <<
                     output_n_triangles << " vs " << max_n_triangles << std::endl;
    }

    return {triangles, triangle_flatten_id};
}


torch::Tensor marching_cubes_dense_cuda(
    torch::Tensor valid_cords,
    torch::Tensor dense_sdf,
    torch::Tensor mask,
    int max_n_triangles
) {
    CHECK_INPUT(valid_cords);
    CHECK_INPUT(dense_sdf);
    CHECK_INPUT(mask);
    assert(max_n_triangles > 0);

    const int num_lif = valid_cords.size(0);

    torch::Tensor triangles = torch::empty({max_n_triangles, 3, 3},
                                           torch::dtype(torch::kFloat32).device(torch::kCUDA));

    dim3 dimBlock = dim3(256);
    uint xBlocks = (num_lif + dimBlock.x - 1) / dimBlock.x;
    dim3 dimGrid = dim3(xBlocks);

    thrust::device_vector<int> n_output(1, 0);
    meshing_cube_dense<<<dimGrid, dimBlock>>>(
        valid_cords.packed_accessor32<int64_t, 2, torch::RestrictPtrTraits>(),
        mask.packed_accessor32<uint8_t, 3, torch::RestrictPtrTraits>(),
        dense_sdf.packed_accessor32<float, 3, torch::RestrictPtrTraits>(),
        triangles.packed_accessor32<float, 3, torch::RestrictPtrTraits>(),
        n_output.data().get(), max_n_triangles
    );

    int output_n_triangles = n_output[0];
    if (output_n_triangles < max_n_triangles) {
        // Trim output tensor if it is not full.
        triangles = triangles.index({torch::indexing::Slice(torch::indexing::None, output_n_triangles)});
    } else {
        // Otherwise spawn a warning.
        std::cerr << "Warning from marching cube: the max triangle number is too small " <<
                     output_n_triangles << " vs " << max_n_triangles << std::endl;
    }

    return triangles;
}
